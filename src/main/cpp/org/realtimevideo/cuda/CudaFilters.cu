// src/main/java/org/realtimevideo/cuda/CudaFilters.cu
#include <hip/hip_runtime.h>
#include <stdio.h> // printf gibi fonksiyonlar için

// CUDA Kernel: Her pikselin ilk bileşenini (BGR formatında Mavi) sıfırlar.
// Eğer RGB ise Kırmızı'yı sıfırlar. OpenCV Mat genellikle BGR'dir.
// BufferedImage TYPE_3BYTE_BGR veya TYPE_INT_RGB olabilir.
// Bu örnekte, her 3 byte'lık grubun ilk byte'ını (B veya R) hedefliyoruz.
__global__ void setFirstChannelToZeroKernel(unsigned char* data, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Görüntü verisi tek boyutlu bir dizi olduğu için pikselin başlangıç indeksini hesapla
        // Her piksel 'channels' kadar byte kaplar (örn: BGR için 3 byte)
        int index = (y * width + x) * channels;
        if (channels >= 3) { // En az 3 kanal varsa (örn: BGR, RGB)
            data[index + 0] = 0; // İlk kanalı (B veya R) sıfırla
                                 // Eğer OpenCV Mat (BGR) ise bu Mavi kanal olur.
                                 // Eğer BufferedImage TYPE_INT_RGB ise ve byte dizisine dönüştürülmüşse, bu Kırmızı olabilir.
                                 // Java tarafındaki veri hazırlığına bağlı.
                                 // Şu anki varsayımımız, Java'dan gelen BytePointer'ın BGR sıralamasında olduğu.
        } else if (channels == 1) { // Gri tonlamalı ise (bu kernel için pek anlamlı değil ama örnek)
            // data[index] = 0; // Gri tonlamalıda tek kanal var, onu sıfırla
        }
    }
}

// Bu fonksiyon Java'dan çağrılacak
extern "C" {
    // Java'daki BytePointer (temelde signed char*) ile eşleşmesi için signed char* kullanıyoruz.
    // Bu pointer hem girdi hem de çıktı verisini taşır.
    void applySetFirstChannelToZeroCuda(signed char* h_input_output_data, int width, int height, int channels) {
        // Kernel ve CUDA bellek işlemleri için unsigned char* kullanacağız, çünkü görüntü verisi genellikle bu şekilde işlenir.
        unsigned char* uc_pixel_data_cpu = reinterpret_cast<unsigned char*>(h_input_output_data);

        unsigned char* d_pixel_data_gpu; // GPU'daki piksel verisi için bellek pointer'ı

        // Bellek boyutunu hesaplarken olası taşmaları önlemek için size_t kullan
        size_t image_size_in_bytes = static_cast<size_t>(width) * height * channels * sizeof(unsigned char);

        hipError_t err; // CUDA API çağrılarının durumunu tutmak için

        // 1. GPU'da bellek ayır
        err = hipMalloc((void**)&d_pixel_data_gpu, image_size_in_bytes);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Malloc Error (%d): %s\n", err, hipGetErrorString(err));
            return; // Hata durumunda fonksiyondan çık
        }

        // 2. Veriyi CPU'dan (uc_pixel_data_cpu) GPU'ya (d_pixel_data_gpu) kopyala
        err = hipMemcpy(d_pixel_data_gpu, uc_pixel_data_cpu, image_size_in_bytes, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Memcpy Host to Device Error (%d): %s\n", err, hipGetErrorString(err));
            hipFree(d_pixel_data_gpu); // Ayrılan GPU belleğini serbest bırak
            return;
        }

        // 3. Kernel'ı başlatmak için grid ve blok boyutlarını ayarla
        // Her blokta 16x16 thread (256 thread) iyi bir başlangıç noktasıdır.
        dim3 threads_per_block(16, 16);
        // Görüntü boyutuna göre gerekli blok sayısını hesapla
        dim3 num_blocks((width + threads_per_block.x - 1) / threads_per_block.x,
                        (height + threads_per_block.y - 1) / threads_per_block.y);

        // 4. Kernel'ı GPU üzerinde çalıştır
        setFirstChannelToZeroKernel<<<num_blocks, threads_per_block>>>(d_pixel_data_gpu, width, height, channels);

        // Kernel çağrısından hemen sonra son hatayı kontrol et (asenkron olabilir)
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Kernel Launch Error (%d): %s\n", err, hipGetErrorString(err));
            hipFree(d_pixel_data_gpu);
            return;
        }

        // Tüm GPU işlemlerinin tamamlanmasını bekle (senkronizasyon)
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Device Sync Error (%d): %s\n", err, hipGetErrorString(err));
            hipFree(d_pixel_data_gpu);
            return;
        }

        // 5. İşlenmiş sonucu GPU'dan (d_pixel_data_gpu) CPU'ya (uc_pixel_data_cpu) geri kopyala
        err = hipMemcpy(uc_pixel_data_cpu, d_pixel_data_gpu, image_size_in_bytes, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "CUDA Memcpy Device to Host Error (%d): %s\n", err, hipGetErrorString(err));
            // Hata olsa bile GPU belleğini serbest bırakmaya çalışacağız.
            // CPU'daki veri bozulmuş olabilir.
        }

        // 6. GPU'da ayrılan belleği serbest bırak
        err = hipFree(d_pixel_data_gpu);
        if (err != hipSuccess) {
            // Bu hata genellikle kritik bir soruna işaret etmez ama loglamak iyi olabilir.
            fprintf(stderr, "CUDA Free Error (%d): %s\n", err, hipGetErrorString(err));
        }
    }
}